#include "hip/hip_runtime.h"
/*
 * device_tensor_gpu.cu
 *
 *  Created on: Mar 2, 2012
 *      Author: tombr
 */

#include "device_tensor.hpp"

namespace tbblas {

template<>
void fft<float, 3, true>(const tensor_base<float, 3, true>& dt, const size_t (&size)[3],
    thrust::device_vector<complex_type<float>::complex_t>& ftdata)
{
  typedef device_tensor<float, 3> tensor_t;
  typedef tensor_proxy<tensor_t::iterator, 3> proxy_t;

  const size_t start[3] = {0, 0, 0};

  tensor_t padded(size);
  proxy_t proxy = subrange(padded, start, dt.size());
  thrust::copy(dt.begin(), dt.end(), proxy.begin());

  hipfftHandle plan;
  hipfftResult res;

  res = hipfftPlan3d(&plan, size[2], size[1], size[0], HIPFFT_R2C);
  hipfftExecR2C(plan, padded.data().data().get(), ftdata.data().get());
  hipfftDestroy(plan);
}

template<>
void ifft<float, 3, true>(thrust::device_vector<complex_type<float>::complex_t>& ftdata,
    const size_t (&size)[3], tensor_base<float, 3, true>& dt)
{
  const size_t count = size[0] * size[1] * size[2];
  typedef device_tensor<float, 3> tensor_t;
  typedef tensor_proxy<tensor_t::const_iterator, 3> const_proxy_t;

  hipfftHandle plan;
  hipfftResult res;

  tensor_t padded(size);
  res = hipfftPlan3d(&plan, size[2], size[1], size[0], HIPFFT_C2R);
  hipfftExecC2R(plan, ftdata.data().get(), padded.data().data().get());
  hipfftDestroy(plan);

  size_t start[3];
  for (int i = 0; i < 3; ++i)
    start[i]= abs((int)size[i] - (int)dt.size()[i]);

  const_proxy_t proxy = subrange(padded / (float)count, start, dt.size());
  thrust::copy(proxy.begin(), proxy.end(), dt.begin());
}

template<>
void fft<double, 3, true>(const tensor_base<double, 3, true>& dt, const size_t (&size)[3],
    thrust::device_vector<complex_type<double>::complex_t>& ftdata)
{
  typedef double value_t;
  typedef device_tensor<value_t, 3> tensor_t;
  typedef tensor_proxy<tensor_t::iterator, 3> proxy_t;

  const size_t start[3] = {0, 0, 0};

  tensor_t padded(size);
  proxy_t proxy = subrange(padded, start, dt.size());
  thrust::copy(dt.begin(), dt.end(), proxy.begin());

  hipfftHandle plan;
  hipfftResult res;

  res = hipfftPlan3d(&plan, size[2], size[1], size[0], HIPFFT_D2Z);
  hipfftExecD2Z(plan, padded.data().data().get(), ftdata.data().get());
  hipfftDestroy(plan);
}

template<>
void ifft<double, 3, true>(thrust::device_vector<complex_type<double>::complex_t>& ftdata,
    const size_t (&size)[3], tensor_base<double, 3, true>& dt)
{
  typedef double value_t;
  typedef device_tensor<value_t, 3> tensor_t;
  typedef tensor_proxy<tensor_t::const_iterator, 3> const_proxy_t;

  const size_t count = size[0] * size[1] * size[2];

  hipfftHandle plan;
  hipfftResult res;

  tensor_t padded(size);
  res = hipfftPlan3d(&plan, size[2], size[1], size[0], HIPFFT_Z2D);
  hipfftExecZ2D(plan, ftdata.data().get(), padded.data().data().get());
  hipfftDestroy(plan);

  size_t start[3];
  for (int i = 0; i < 3; ++i)
    start[i]= abs((int)size[i] - (int)dt.size()[i]);

  const_proxy_t proxy = subrange(padded / (value_t)count, start, dt.size());
  thrust::copy(proxy.begin(), proxy.end(), dt.begin());
}

}
