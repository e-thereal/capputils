#include "hip/hip_runtime.h"
/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
/* Includes, cuda */
#include <hipblas.h>
#include <cutil.h>
#include <hip/hip_runtime.h>

/* Includes, project */
#include "framework.h"
#include "deviceSelect.h"
#include "svmClassify.h"
#include "svmClassifyKernels.h"


namespace gpusvm {

/**
 * Performs SVM classification.
 * @param data the data to be classfied, stored as a flat column major array.
 * @param nData the number of data points being classified
 * @param supportVectors the support vectors of the classifier, stored as a flat column major array.
 * @param nSV the number of support vectors of the classifier
 * @param nDimension the dimensionality of the data and support vectors
 * @param kp a struct containing all the information about the kernel parameters
 * @param p_result a pointer to a float pointer where the results will be placed.  The perform classification routine will allocate the output buffer.
 */
void performClassification(float *data, int nData, float *supportVectors, int nSV, int nDimension, float* alpha, Kernel_params kp, float** p_result)
{
  chooseLargestGPU(true);
	int total_nPoints = nData;
	int nPoints;	
	float gamma,coef0,b;
	int degree;
	
	if(kp.kernel_type.compare(0,3,"rbf") == 0)
	{
		printf("Found RBF kernel\n");
		gamma=kp.gamma;
		b=kp.b;
	}
	else if(kp.kernel_type.compare(0,10,"polynomial") == 0)
	{
		printf("Found polynomial kernel\n");
		gamma=kp.gamma;
		degree=kp.degree;
		coef0 = kp.coef0;
		b=kp.b;
	}
	else if(kp.kernel_type.compare(0,6,"linear") == 0)
	{
		printf("Found linear kernel\n");
		gamma = 1.0;
		b=kp.b;
	}
	else if(kp.kernel_type.compare(0,7,"sigmoid") == 0)
	{
		printf("Found sigmoid kernel\n");
		gamma = kp.gamma;
		coef0 = kp.coef0;
		b=kp.b;
		//printf("gamma = %f coef0=%f\n",gamma, coef0);
	}
	else
	{
		printf("Error: Unknown kernel type - %s\n",kp.kernel_type.c_str());
		exit(0);
	}
	
	int nBlocksSV = intDivideRoundUp(nSV,GPUSVM_BLOCKSIZE);

//	hipblasStatus_t status = cublasInit();
//	if (status != HIPBLAS_STATUS_SUCCESS) {
//		printf("CUBLAS initialization error\n");
//		exit(0);
//	}


	float* devSV;
	size_t devSVPitch;
	CUDA_SAFE_CALL(hipMallocPitch((void**)&devSV, &devSVPitch, nSV*sizeof(float), nDimension));
	CUDA_SAFE_CALL(hipMemcpy2D(devSV, devSVPitch, supportVectors, nSV*sizeof(float), nSV*sizeof(float), nDimension, hipMemcpyHostToDevice));
	int devSVPitchInFloats = ((int)devSVPitch) / sizeof(float);


	float* devAlpha;
	CUDA_SAFE_CALL(hipMalloc((void**)&devAlpha, nSV*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpy(devAlpha, alpha, nSV*sizeof(float), hipMemcpyHostToDevice));
	
	


	float* devLocalValue;

	float* devResult;
	
	float* result = (float*)malloc(total_nPoints*sizeof(float));
	*(p_result) = result;

	float* devSVDots;
	CUDA_SAFE_CALL(hipMalloc((void**)&devSVDots, sizeof(float)*nSV));


	size_t free_memory,total_memory;
	hipMemGetInfo(&free_memory,&total_memory);
	//printf("\nChecking GPU Memory status...\n");
	//printf("Total Memory=%d bytes   Available Memory=%d bytes\n",total_memory, free_memory);
	size_t free_memory_floats = free_memory/sizeof(float);
	
	free_memory_floats = (size_t)(0.9 * free_memory_floats); 

	nPoints = ((free_memory_floats-devSVPitchInFloats*nDimension-nSV-nSV)/(nDimension+1+devSVPitchInFloats+1+nBlocksSV));
	nPoints = (nPoints>>7)<<7;		//for pitch limitations assigning to be a multiple of 128
	
	nPoints = min(nPoints, total_nPoints); //for few points
	nPoints = min(nPoints, (int)GPUSVM_MAX_POINTS); //for too many points

	//printf("Max points that can reside in GPU memory per call = %d\n\n", nPoints);
	
	dim3 mapGrid(intDivideRoundUp(nSV, GPUSVM_BLOCKSIZE), nPoints);
	dim3 mapBlock(GPUSVM_BLOCKSIZE);
	

	dim3 reduceGrid(1, nPoints);
	dim3 reduceBlock(mapGrid.x, 1);


	float* devData;
	size_t devDataPitch;
	CUDA_SAFE_CALL(hipMallocPitch((void**)&devData, &devDataPitch, nPoints*sizeof(float), nDimension));

	int devDataPitchInFloats = ((int)devDataPitch) / sizeof(float);

	float* devDataDots;
	CUDA_SAFE_CALL(hipMalloc((void**)&devDataDots, sizeof(float)*nPoints));

	CUDA_SAFE_CALL(hipMalloc((void**)&devLocalValue, sizeof(float)*mapGrid.x*mapGrid.y));
	
	CUDA_SAFE_CALL(hipMalloc((void**)&devResult, sizeof(float)*mapGrid.y));
	
	float* devDots;
	size_t devDotsPitch;
	CUDA_SAFE_CALL(hipMallocPitch((void**)&devDots, &devDotsPitch, nSV*sizeof(float), nPoints));


		
	
	dim3 threadsLinear(GPUSVM_BLOCKSIZE);
	if(kp.kernel_type.compare(0,3,"rbf")==0)
	{
		dim3 blocksSVLinear(intDivideRoundUp(nSV, GPUSVM_BLOCKSIZE));
		makeSelfDots<<<blocksSVLinear, threadsLinear>>>(devSV, devSVPitchInFloats, devSVDots, nSV, nDimension);
	}

	int iteration=1;

	for(int dataoffset=0; dataoffset<total_nPoints; dataoffset += nPoints) 
	{
		// code for copying data
		if(dataoffset+nPoints > total_nPoints)
		{
			nPoints = total_nPoints-dataoffset;
			mapGrid=dim3(intDivideRoundUp(nSV, GPUSVM_BLOCKSIZE), nPoints);
			mapBlock=dim3(GPUSVM_BLOCKSIZE);
	
			reduceGrid=dim3(1, nPoints);
			reduceBlock=dim3(mapGrid.x, 1);

			CUDA_SAFE_CALL(hipFree(devLocalValue));
			CUDA_SAFE_CALL(hipMalloc((void**)&devLocalValue, sizeof(float)*mapGrid.x*mapGrid.y));
	
			//resize & copy devdata, devdots,
			CUDA_SAFE_CALL(hipFree(devData));
			CUDA_SAFE_CALL(hipMallocPitch((void**)&devData, &devDataPitch, nPoints*sizeof(float), nDimension));
			devDataPitchInFloats = devDataPitch/sizeof(float);
		}
		
		//printf("Number of Points in call #%d=%d \n",iteration, nPoints);
		
		if(total_nPoints*sizeof(float) < GPUSVM_MAX_PITCH)
		{	
			CUDA_SAFE_CALL(hipMemcpy2D(devData, devDataPitch, data+dataoffset, total_nPoints*sizeof(float), nPoints*sizeof(float), nDimension, hipMemcpyHostToDevice));
		}
		else
		{
			for(int nd=0;nd<nDimension;nd++)
			{
				CUDA_SAFE_CALL(hipMemcpy(devData+nd*devDataPitchInFloats, data+nd*total_nPoints+dataoffset, nPoints*sizeof(float), hipMemcpyHostToDevice));	
			}
		}

		dim3 blocksDataLinear(intDivideRoundUp(nPoints, GPUSVM_BLOCKSIZE));
		dim3 threadsDots(GPUSVM_BLOCKSIZE, 1);
		dim3 blocksDots(intDivideRoundUp(nSV, GPUSVM_BLOCKSIZE), intDivideRoundUp(nPoints, GPUSVM_BLOCKSIZE));
		int devDotsPitchInFloats = ((int)devDotsPitch)/ sizeof(float);
	
		if(kp.kernel_type.compare(0,3,"rbf")==0)
		{
			makeSelfDots<<<blocksDataLinear, threadsLinear>>>(devData, devDataPitchInFloats, devDataDots, nPoints, nDimension);
		
			CUDA_SAFE_CALL(hipMemset(devDots, 0, sizeof(float)*devDotsPitchInFloats*nPoints));

			makeDots<<<blocksDots, threadsDots>>>(devDots, devDotsPitchInFloats, devSVDots, devDataDots, nSV, nPoints);
	
			hipDeviceSynchronize(); //unnecessary..onyl for timing..
		}

		float sgemmAlpha, sgemmBeta;
		if(kp.kernel_type.compare(0,3,"rbf") == 0)
		{
			sgemmAlpha = 2*gamma;
			sgemmBeta = -gamma;
		}
		else
		{
			sgemmAlpha = gamma;
			sgemmBeta = 0.0f;
		}

		hipblasSgemm('n', 't', nSV, nPoints, nDimension, sgemmAlpha, devSV, devSVPitchInFloats, devData, devDataPitchInFloats, sgemmBeta, devDots, devDotsPitchInFloats);

		hipDeviceSynchronize();

		int reduceOffset = (int)pow(2, ceil(log2((float)GPUSVM_BLOCKSIZE))-1);
		//printf("size: %d -> reduceOffset: %d\n", BLOCKSIZE, reduceOffset);
		int sharedSize = sizeof(float)*(GPUSVM_BLOCKSIZE);

    
		if(kp.kernel_type.compare(0,3,"rbf") == 0)
		{
			computeKernelsReduce<<<mapGrid, mapBlock, sharedSize>>>(devDots, devDotsPitchInFloats, devAlpha, nPoints, nSV, GPUSVM_RBF, 0,1, devLocalValue, 1<<int(ceil(log2((float)GPUSVM_BLOCKSIZE))-1));
		}
		else if(kp.kernel_type.compare(0,10,"polynomial") == 0)
		{
			computeKernelsReduce<<<mapGrid, mapBlock, sharedSize>>>(devDots, devDotsPitchInFloats, devAlpha, nPoints, nSV, GPUSVM_POLYNOMIAL, coef0, degree, devLocalValue, 1<<int(ceil(log2((float)GPUSVM_BLOCKSIZE))-1));
		}
		else if(kp.kernel_type.compare(0,6,"linear") == 0)
		{
			computeKernelsReduce<<<mapGrid, mapBlock, sharedSize>>>(devDots, devDotsPitchInFloats, devAlpha, nPoints, nSV, GPUSVM_LINEAR, 0,1, devLocalValue, 1<<int(ceil(log2((float)GPUSVM_BLOCKSIZE))-1));
		}
		else if(kp.kernel_type.compare(0,7,"sigmoid") == 0)
		{
			computeKernelsReduce<<<mapGrid, mapBlock, sharedSize>>>(devDots, devDotsPitchInFloats, devAlpha, nPoints, nSV, GPUSVM_SIGMOID, coef0, 1, devLocalValue, 1<<int(ceil(log2((float)GPUSVM_BLOCKSIZE))-1));
		}

	
		reduceOffset = (int)pow(2, ceil(log2((float)mapGrid.x))-1);
		sharedSize = sizeof(float)*mapGrid.x;

		doClassification<<<reduceGrid, reduceBlock, sharedSize>>>(devResult, b, devLocalValue, reduceOffset, mapGrid.x);
	
		hipDeviceSynchronize(); //unnecessary..onyl for timing..
	
		//printf("rest of stuff = %f\n",blas1time+(float)((f.tv_sec-s.tv_sec)+(f.tv_usec-s.tv_usec)/1e6));
	
		hipMemcpy(result+dataoffset, devResult, nPoints*sizeof(float), hipMemcpyDeviceToHost);


		iteration++;
	}
	
	
	CUDA_SAFE_CALL(hipFree(devResult));
	CUDA_SAFE_CALL(hipFree(devAlpha));
	CUDA_SAFE_CALL(hipFree(devData));
	CUDA_SAFE_CALL(hipFree(devLocalValue));
	CUDA_SAFE_CALL(hipFree(devDots));
	CUDA_SAFE_CALL(hipFree(devSV));
	CUDA_SAFE_CALL(hipFree(devSVDots));
	CUDA_SAFE_CALL(hipFree(devDataDots));

}

}
